#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "cuda_utils.cuh"



// basic math op kernels ensure 4|A| = |C| and offset <= 3|A|
__device__ void gpu_add(int* a, int* b, int *c, int offset){
    const int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    c[offset + tid] = a[tid] + b[tid];
}
__device__ void gpu_sub(int* a, int* b, int *c, int offset){
    const int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    c[offset + tid] = a[tid] - b[tid];
}
__device__ void gpu_mul(int* a, int* b, int *c, int offset){
    const int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    c[offset + tid] = a[tid] * b[tid];
}
__device__ void gpu_xor(int* a, int* b, int *c, int offset){
    const int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    c[offset + tid] = a[tid] ^ b[tid];
}
    
__global__ void mplex_kernel(int ksel, int *a, int *b, int *c, int offset){
    switch(ksel) {
        case 0: {
            gpu_add(a, b, c, offset);
            break;
        } case 1: {
            gpu_sub(a, b, c, offset);
            break;
        } case 2: {
            gpu_mul(a, b, c, offset);
            break;
        } case 3: {
            gpu_xor(a, b, c, offset);
            break;
        }
    }
}


void printResultsSync(int N, int* h_c, float t, int idx)
{
    printf("Sync kernels finished in %f ms", t);
    printf("A[%d] + B[%d] = %d \n", idx, idx, h_c[idx]);
    printf("A[%d] - B[%d] = %d \n", idx, idx, h_c[idx+N]);
    printf("A[%d] * B[%d] = %d \n", idx, idx, h_c[idx+N*2]);
    printf("A[%d] ^ B[%d] = %d \n", idx, idx, h_c[idx+N*3]);
}

void testSync(int N, int blockSize, int numBlocks, int testIdx,
        int *h_a, int *h_b,int *h_c,
        int *d_a, int *d_b, int *d_c)
{
    hipEvent_t start, stop;
    checkCuda( hipEventCreate(&start) );
    checkCuda( hipEventCreate(&stop) );

    printf("SyncTest start\n");
    checkCuda( hipEventRecord(start, 0) );
    // simulate new data coming in for parity
    for(int i = 0; i < 4; i++) 
    {
        printf("Memcpy input\n");
        checkCuda( hipMemcpy(d_a, h_a, N*sizeof(int), hipMemcpyHostToDevice) );
        checkCuda( hipMemcpy(d_b, h_b, N*sizeof(int), hipMemcpyHostToDevice) );
        printf("Kernel %d exec\n", i); 
        checkCudaKernel( (mplex_kernel<<<N, blockSize>>>(i, d_a, d_b, d_c, N)) );
        printf("Memcpy result\n");
        checkCuda( hipMemcpy(&h_c[N*i], &d_c[N*i], N*sizeof(int), hipMemcpyDeviceToHost) );
    }

    printf("Kernels launched");
 
    float t;
    checkCuda( hipEventRecord(stop, 0) );
    checkCuda( hipEventElapsedTime(&t, start, stop));
    printResultsSync(N, h_c, t, testIdx);


}
    
void printResultsStream(int N, int* h_c, float t, int idx){
    
    printf("Stream kernels finished in %f ms", t);
    printf("A[%d] + B[%d] = %d \n", idx, idx, h_c[idx]);
    printf("A[%d] - B[%d] = %d \n", idx, idx, h_c[idx+N]);
    printf("A[%d] * B[%d] = %d \n", idx, idx, h_c[idx+N*2]);
    printf("A[%d] ^ B[%d] = %d \n", idx, idx, h_c[idx+N*3]);
}

void testStream(int N, int blockSize, int numBlocks, int testIdx,
        int *h_a, int *h_b, int *h_c,
        int *d_a, int *d_b, int *d_c)
{

    hipEvent_t start, end;
    checkCuda( hipEventCreate(&start) );
    checkCuda( hipEventCreate(&end) );
    
    printf("Starting streaming approach \n");
    checkCuda( hipEventRecord(start, 0));

    hipStream_t streams[4]; // lets running everything in parallel
    for(int i = 0; i < 4; i++) {
        checkCuda( hipStreamCreate(&streams[i]) );
    }

    for(int i = 0; i < 4; i++){
        checkCuda( hipMemcpyAsync(d_a, h_a, sizeof(int) * N, hipMemcpyHostToDevice, streams[i]) );
        checkCuda( hipMemcpyAsync(d_b, h_b, sizeof(int) * N, hipMemcpyHostToDevice, streams[i]) );
        checkCudaKernel( (mplex_kernel<<<N, blockSize, 0, streams[i]>>>(i, d_a, d_b, d_c, 0)) );
        checkCuda( hipMemcpyAsync(&h_c[N*i], &d_c[N*i], sizeof(int)*N, hipMemcpyHostToDevice, streams[i]) );
    }

    for(int i = 0; i < 4; i++) {
        checkCuda( hipStreamSynchronize(streams[i]) ); // sync all threads
    }
    
    checkCuda( hipEventRecord(end, 0) ); 
    float t;
    checkCuda( hipEventElapsedTime(&t, start, end));
    printResultsStream(N, h_c, t, testIdx);


}


// print the specs of this machine and number of devices
void printDeviceSpecs(){
    
    hipDeviceProp_t prop;
    checkCuda( hipGetDeviceProperties(&prop, 0) );
    printf("Cuda Device %s\n", prop.name);

    int numDevices;
    checkCuda( hipGetDeviceCount(&numDevices) );
    printf("you have %d devices\n", numDevices);
}

void allocateData(int N, int **h_a, int **h_b, int **h_c, int **d_a, int **d_b, int **d_c)
{
    checkCuda( hipHostMalloc((void **)h_a, sizeof(int) * N) );
    checkCuda( hipHostMalloc((void **)h_b, sizeof(int) * N) );
    checkCuda( hipHostMalloc((void **)h_c, sizeof(int) * 4 *N) );
    checkCuda( hipMalloc((void**) d_a, sizeof(int) * 4 * N) ); // we need more space on recv
    checkCuda( hipMalloc((void**) d_b, sizeof(int) * 4 * N) );
    checkCuda( hipMalloc((void**) d_c, sizeof(int) * 4 * N) );
    
    // dummy data input
    for(int i = 0; i < N; i++){
        (*h_a)[i] = rand() %10;
        (*h_b)[i] = rand() %10;
    }
}

void freeData(int *h_a, int *h_b, int *h_c, int *d_a, int *d_b, int *d_c)
{
    checkCuda( hipHostFree(h_a) );
    checkCuda( hipHostFree(h_b) );
    checkCuda( hipHostFree(h_c) );
    checkCuda( hipFree(d_a) );
    checkCuda( hipFree(d_b) );
    checkCuda( hipFree(d_c) );
}

int main(int argc, char** argv)
{
	// read command line arguments
    int N = (1 << 20);
	int blockSize = 256;
	
	if (argc >= 2) {
		N = atoi(argv[1]);
	}
	if (argc >= 3) {
		blockSize = atoi(argv[2]);
	}
	int numBlocks = N/blockSize;

	// validate command line arguments
	if (N % blockSize != 0) {
		++numBlocks;
		N = numBlocks*blockSize;	
	}

    printDeviceSpecs(); 
    
    int *h_a, *h_b, *h_c;  // C is 4x len(a) 
    int *d_a, *d_b, *d_c;

    srand(time(NULL));
    int testIdx = rand() % N;
    printf("Allocating data\n");
    allocateData(N, &h_a, &h_b, &h_c, &d_a, &d_b, &d_c);
    printf("Allocating done running kernels\n");

    testSync(N, blockSize, numBlocks, testIdx, h_a, h_b, h_c, d_a, d_b, d_c);
    testStream(N, blockSize, numBlocks, testIdx, h_a, h_b, h_c, d_a, d_b, d_c); 

    printf("Free data\n");
    freeData(h_a, h_b, h_c, d_a, d_b, d_c);
    
}
