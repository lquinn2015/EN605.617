#include "cuda_utils.cuh"
#include "hipfft/hipfft.h"


hipError_t checkCuda_z(hipError_t result, char const* file, int line)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n in %s:%d",
            hipGetErrorString(result), file, line);
    assert(result == hipSuccess);
  }
    #ifdef DEBUG  
  fprintf(stdout,"cuda result %d\n", result);
    #endif
  return result;
}

hipfftResult_t checkCufft_z(hipfftResult_t result, char const* file, int line)
{
  if (result != HIPFFT_SUCCESS) {
    fprintf(stderr, "cufft Runtime Error: \n in %s:%d",
            file, line);
    assert(result == HIPFFT_SUCCESS);
  }
    #ifdef DEBUG  
  fprintf(stdout,"cuFFT result %d\n", result);
    #endif
  return result;
}
