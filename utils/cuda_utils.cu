#include "cuda_utils.cuh"
#include "hipfft/hipfft.h"



hipError_t checkCuda_z(hipError_t result, char* file, int line)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n in %s:%d",
            hipGetErrorString(result), file, line);
    assert(result == hipSuccess);
  }
  return result;
}


hipfftResult_t checkCufft_z(hipfftResult_t result, char* file, int line)
{
  if (result != HIPFFT_SUCCESS) {
    fprintf(stderr, "cufft Runtime Error: \n in %s:%d",
            file, line);
    assert(result == hipSuccess);
  }
  return result;
}
