#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_complex.h"
#include "cuda_utils.cuh"
#include <hipfft/hipfft.h>

// globals
static FILE* gnuplot;

__global__ void fft2amp(int n, hipFloatComplex *fft, float *db){

    const int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    int idx = tid;
    while( idx < n){
        db[idx] = hipCabsf(fft[idx]);
        idx += tid;
    }

}

hipFloatComplex* readData(int *n, double *&idata, double *&qdata)
{
    // IQ data from FMcapture1.dat
    FILE* f = fopen("FMcapture1.dat", "r");
    fseek(f, 0, SEEK_END);
    int samples = ftell(f) / 2; // IQ samples are 8 bit unsigned  values
    rewind(f);
    unsigned char* data = (unsigned char*) malloc(2*samples * sizeof(char));
    idata  = (double*) malloc(samples * sizeof(double));
    qdata  = (double*) malloc(samples * sizeof(double));
    fread(data, 1, samples*2, f);
    hipFloatComplex *z = (hipFloatComplex *) malloc(sizeof(hipFloatComplex) * samples);
    
    for(int i = 0; i < samples; i++){
        z[i] = make_hipFloatComplex( (float)data[2*i] - 127.0, (float)data[2*i+1] -127.0 );
        idata[i] = data[2*i] - 127.0;
        qdata[i] = data[2*i+1] - 127.0;
    
    }
    free(data); 
    fclose(f);
    *n = samples;
    return z;
}
static int cplot = 0;
void plot_xy_data(double* x, double *y, int n)
{
    fprintf(gnuplot, "set term wxt %d size 500,500\n", cplot++ );
    fprintf(gnuplot, "plot '-' \n");

    for(int i = 0; i < n; i++){
        fprintf(gnuplot,"%lf, %lf\n", x[i], y[i]);
    }
    fprintf(gnuplot, "e\n");
}

    
// user job to insure that z[offset+n] does not overboubd 
void create_fft(hipFloatComplex *z, int n, int offset, hipStream_t s){
    
    hipfftComplex *d_sig, *d_fft;
    float * d_db; 
    checkCuda( hipMalloc((void**)&d_sig, sizeof(hipfftComplex) * n) );
    checkCuda( hipMalloc((void**)&d_fft, sizeof(hipfftComplex) * n) );
    checkCuda( hipMalloc((void**)&d_db, sizeof(float) * n) );

    checkCuda( hipMemcpyAsync(d_sig, &z[offset], n*sizeof(hipfftComplex), hipMemcpyHostToDevice, s) );
    
    hipfftHandle plan;
    checkCufft( hipfftPlan1d(&plan, n, HIPFFT_C2C, 1) ); // issuing 1 FFT of the size sample
    checkCufft( hipfftSetStream(plan, s) );
    checkCufft( hipfftExecC2C(plan, d_sig, d_fft, HIPFFT_FORWARD) ); // execute the plan

    // we have a FFT we need to extract and plot the amplitude of it now
    checkCudaKernel( (fft2amp<<<1, 1024, 0, s>>>(n, d_fft, d_db)) );
    float * db = (float*) malloc(n*sizeof(float)); 
    checkCuda( hipMemcpyAsync(db, d_db, n*sizeof(float), hipMemcpyDeviceToHost, s) );
    

    checkCuda( hipStreamSynchronize(s) );

    fprintf(gnuplot, "plot '-' smooth frequency with linespoints lt -1 notitle");
    for(int i = 0; i < n; i++){
        fprintf(gnuplot,"%d  %f\n", i, db[i]);
    }
    fprintf(gnuplot, "e\n");
    
    checkCufft( hipfftDestroy(plan) );
    checkCuda( hipFree(d_sig) );
    checkCuda( hipFree(d_fft) );
    free(db);

}


int main(int argc, char** argv)
{
    int n;
    double *idata, *qdata;
    hipFloatComplex *z = readData(&n, idata, qdata); // we have n complex numbers now

    gnuplot = popen("gnuplot -persistent", "w");
    
       

    

}
