#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_complex.h"
#include "cuda_utils.cuh"
#include <hipfft/hipfft.h>

// globals
static FILE* gnuplot;

__global__ void fft2amp(int n, hipFloatComplex *fft, float *db){

    const int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    int idx = tid;
    while( idx < n){
        db[idx] = log10(hipCabsf(fft[idx]));
        idx += blockDim.x;
    }

}

hipFloatComplex* readData(int *n, double *&idata, double *&qdata)
{
    // IQ data from FMcapture1.dat
    FILE* f = fopen("FMcapture1.dat", "r");
    fseek(f, 0, SEEK_END);
    int samples = ftell(f) / 2; // IQ samples are 8 bit unsigned  values
    rewind(f);
    unsigned char* data = (unsigned char*) malloc(2*samples * sizeof(char));
    idata  = (double*) malloc(samples * sizeof(double));
    qdata  = (double*) malloc(samples * sizeof(double));
    fread(data, 1, samples*2, f);
    hipFloatComplex *z = (hipFloatComplex *) malloc(sizeof(hipFloatComplex) * samples);
    
    for(int i = 0; i < samples; i++){
        z[i] = make_hipFloatComplex( (float)data[2*i] - 127.0, (float)data[2*i+1] -127.0 );
        idata[i] = data[2*i] - 127.0;
        qdata[i] = data[2*i+1] - 127.0;
    
    }
    free(data); 
    fclose(f);
    *n = samples;
    return z;
}
static int cplot = 0;
void plot_xy_data(double* x, double *y, int n)
{
    fprintf(gnuplot, "set term wxt %d size 500,500\n", cplot++ );
    fprintf(gnuplot, "plot '-' \n");

    for(int i = 0; i < n; i++){
        fprintf(gnuplot,"%lf, %lf\n", x[i], y[i]);
    }
    fprintf(gnuplot, "e\n");
}

    
// user job to insure that z[offset+n] does not overboubd 
void create_fft(hipFloatComplex *z, int n, int offset, hipStream_t s,
    float f_c, // freqency center 
    float f_s  // sample rate 
){
    
    printf("Starting FFT\n");
    hipfftComplex *d_sig, *d_fft;
    float * d_db; 
    checkCuda( hipMalloc((void**)&d_sig, sizeof(hipfftComplex) * n) );
    checkCuda( hipMalloc((void**)&d_fft, sizeof(hipfftComplex) * n) );
    checkCuda( hipMalloc((void**)&d_db, sizeof(float) * n) );

    checkCuda( hipMemcpyAsync(d_sig, &z[offset], n*sizeof(hipfftComplex), hipMemcpyHostToDevice, s) );
    
    printf("Starting plan\n");
    hipfftHandle plan;
    checkCufft( hipfftPlan1d(&plan, n, HIPFFT_C2C, 1) ); // issuing 1 FFT of the size sample
    checkCufft( hipfftSetStream(plan, s) );
    checkCufft( hipfftExecC2C(plan, d_sig, d_fft, HIPFFT_FORWARD) ); // execute the plan

    printf("Starting kernel\n");
    // we have a FFT we need to extract and plot the amplitude of it now
    checkCudaKernel( (fft2amp<<<1, 1024, 0, s>>>(n, d_fft, d_db)) );
    float * db = (float*) malloc(n*sizeof(float)); 
    // db is display as  0,1,2..Fs/2 -Fs/2 ... -3 -2. -1 reorder it 
    checkCuda( hipMemcpyAsync(db, &d_db[n/2], n/2*sizeof(float), hipMemcpyDeviceToHost, s) );
    
    checkCuda( hipMemcpyAsync(&db[n/2], &d_db[n/2], n/2*sizeof(float), hipMemcpyDeviceToHost, s) );

    printf("Sync start\n");
    checkCuda( hipStreamSynchronize(s) );
    printf("Sync Complete ploting now\n");

    float Fc_Mhz = f_c / 1e6; // div by 10^6 to shift to mhz units
    float Fs_Mhz = f_s / 1e6;

    float lowF = Fc_Mhz - Fs_Mhz; 
    float highF = Fc_Mhz + Fs_Mhz;

    fprintf(gnuplot, "set xtics ('%.1f' 1, '%.1f' %d, '%1.f' %d)\n", lowF, Fc_Mhz, n/2, highF, n-1);
    fprintf(gnuplot, "plot '-' smooth frequency with linespoints lt -1 notitle\n");
    for(int i = 0; i < n; i++){
        fprintf(gnuplot,"%d  %f\n", i, db[i]);
    }
    fprintf(gnuplot, "e\n");
    
    checkCufft( hipfftDestroy(plan) );
    checkCuda( hipFree(d_sig) );
    checkCuda( hipFree(d_fft) );
    free(db);
    printf("Finish fft\n");

}


int main(int argc, char** argv)
{
    int n;
    double *idata, *qdata;
    hipFloatComplex *z = readData(&n, idata, qdata); // we have n complex numbers now

    #ifdef DPLOT
    gnuplot = popen("gnuplot -persistent", "w");
    #else
    gnuplot = fopen("gplot", "w");    
    #endif

    hipStream_t s;
    checkCuda( hipStreamCreate(&s));
    create_fft(z, 5000, 0, s, 100.122e6, 2.5e6);
    
    

}
