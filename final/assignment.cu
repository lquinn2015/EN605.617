#include "hip/hip_runtime.h"
#include "assignment.cuh" // important globals are defined here read it


__global__ void freqShift(int n, hipFloatComplex *S, 
     float shiftF, float intialPhase, float sampleF)
{
    const int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    unsigned int stride = gridDim.x * blockDim.x;
    unsigned int idx = tid;

    float dt = shiftF / sampleF;

    while(idx < n)
    {

        float f_t = dt * (float)idx;
        hipFloatComplex shiftVec = make_hipFloatComplex(cospif(2*f_t), sinpif(2*f_t)); 
        S[idx] = hipCmulf(shiftVec, S[idx]);

        idx += stride;
    }

}

__global__ void pdsC2R(int n, hipFloatComplex *sig, float *r)
{
    const int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    unsigned int stride = gridDim.x * blockDim.x;
    unsigned int idx = tid;

    while(idx < n) {
        
        if(idx+1 > n) break;
        
        hipFloatComplex p = hipCmulf(hipConjf(sig[idx]), sig[idx+1]);
        r[idx] = atan( hipCimagf(p) / hipCrealf(p)); // atan( Im(p) / Re(p))
        idx += stride;
    }
 
}
__global__ void decimateC2C(int n, int dec, hipFloatComplex *S, 
        hipFloatComplex *R)
{

    const int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    int strideD = stride * dec;
    int idx = tid;
    int idxD = tid * dec;
    while(idxD < n) 
    {
        R[idx] = S[idxD];
        idx += stride;
        idxD += strideD;
    } 
}

__global__ void decimateR2R(int n, int d, float *S, float *R)
{
    const int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    int stride = gridDim.x * blockDim.x * d;
    int strideD = stride * d;
    int idx = tid;
    int idxD = tid * d;
    while(idxD < n) 
    {
        R[idx] = S[idxD];
        idx += stride;
        idxD += strideD;
    } 
}

__global__ void scaleVec(int n, float *s, float normal)
{
    const int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    int idx = tid;

    while(idx < n)
    {
        s[idx] = 10000 * s[idx] / normal;
        idx += stride;
    }
}

__global__ void blackmanFIR_200KHz( int n, hipFloatComplex *S,
                                           hipFloatComplex *R)
{
    const int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    int idx = tid;

    while(idx < n)
    {
        float I = 0;
        float Q = 0; 
        for(int k = 0; k < c_BLACKMAN_LPF_200KHz_len; k++)
        {
            if(idx-k < 0 ) continue; // ease of impl ignore lower samples
            hipFloatComplex F = S[idx-k];
            I += c_BLACKMAN_LPF_200KHz[k] * hipCrealf(F);
            Q += c_BLACKMAN_LPF_200KHz[k] * hipCimagf(F);
        }

        R[idx] = make_hipFloatComplex(I, Q);
        idx += stride; 
    }
}

__global__ void findMaxR2RMag(int n, float *arr,  float *db){
    
    __shared__ float cache[c_FIND_MAX_CACHESIZE];
    unsigned idx = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned stride = gridDim.x * blockDim.x;
    unsigned offset = 0;
    
    float *max = &db[n]; // db has a max at n
    int* mutex = (int*) &db[n+1]; // and lock at 0;
    
    float tmp = -1.0;
    while(idx + offset < n){
        tmp = fmaxf(tmp, abs(arr[idx+offset]));
        offset += stride;
    }
    cache[threadIdx.x] = tmp;
    __syncthreads();

    //reduce in the block
    unsigned int i = blockDim.x/2; 
    while(i != 0){
        if(threadIdx.x < i){
            cache[threadIdx.x] = fmaxf(cache[threadIdx.x], cache[threadIdx.x+i]);
        }
        __syncthreads();
        i /= 2;
    }
    // reduce among all blocks
    if(threadIdx.x == 0){
        while(atomicCAS(mutex, 0, 1) != 0); // lock
        *max =fmaxf(*max, cache[0]);
        atomicExch(mutex, 0); // unlock
    }

}


__global__ void findMaxC2RMag(int n, hipFloatComplex *arr,  float *db)
{
    //assert(c_FIND_MAX_CACHESIZE >= blockDim.x);
    __shared__ float cache[c_FIND_MAX_CACHESIZE];

    unsigned idx = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned stride = gridDim.x * blockDim.x;
    unsigned offset = 0;
    
    float *max = &db[n]; // db has a max at n
    int* mutex = (int*) &db[n+1]; // and lock at 0;
    
    float tmp = -1.0;
    while(idx + offset < n){
        tmp = fmaxf(tmp, hipCabsf(arr[idx+offset]));
        offset += stride;
    }
    cache[threadIdx.x] = tmp;
    __syncthreads();

    //reduce in the block
    unsigned int i = blockDim.x/2; 
    while(i != 0){
        if(threadIdx.x < i){
            cache[threadIdx.x] = fmaxf(cache[threadIdx.x], cache[threadIdx.x+i]);
        }
        __syncthreads();
        i /= 2;
    }
    // reduce among all blocks
    if(threadIdx.x == 0){
        while(atomicCAS(mutex, 0, 1) != 0); // lock
        *max =fmaxf(*max, cache[0]);
        atomicExch(mutex, 0); // unlock
    }
}

__global__ void fft2amp(int n, hipFloatComplex *fft, float *db)
{
    float dbMax = db[n];
    const int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    unsigned stride = gridDim.x * blockDim.x;
    
    int idx = tid;
    while( idx < n){
        db[idx] = c_dBAdjustment * log10(hipCabsf(fft[idx])/dbMax) ;
        idx += stride; 
    }
}

hipFloatComplex* readData(int *n, double *&idata, double *&qdata)
{
    // IQ data from FMcapture1.dat
    FILE* f = fopen("FMcapture1.dat", "r");
    fseek(f, 0, SEEK_END);
    int samples = ftell(f) / 2; // IQ samples are 8 bit unsigned  values
    rewind(f);
    unsigned char* data = (unsigned char*) malloc(2*samples * sizeof(char));
    idata  = (double*) malloc(samples * sizeof(double));
    qdata  = (double*) malloc(samples * sizeof(double));
    fread(data, 1, samples*2, f);
    hipFloatComplex *z = (hipFloatComplex *) malloc(sizeof(hipFloatComplex) * samples);
    
    for(int i = 0; i < samples; i++){
        z[i] = make_hipFloatComplex( (float)data[2*i] - 127.0, (float)data[2*i+1] -127.0 );
        idata[i] = data[2*i] - 127.0;
        qdata[i] = data[2*i+1] - 127.0;
    }
    free(data); 
    fclose(f);
    *n = samples;
    return z;
}

void plot_xy_data(double* x, double *y, int n)
{
    fprintf(gnuplot, "set term wxt %d size 500,500\n", cplot++ );
    fprintf(gnuplot, "plot '-' \n");

    for(int i = 0; i < n; i++){
        fprintf(gnuplot,"%lf, %lf\n", x[i], y[i]);
    }
    fprintf(gnuplot, "e\n");
}

void plotfft(float f_c, float f_s, int n, float* db, const char* title){

    float Fc_Mhz = f_c / 1e6; // div by 10^6 to shift to mhz units
    float Fs_Mhz = f_s / 1e6;

    float lowF = Fc_Mhz - Fs_Mhz/2; 
    float highF = Fc_Mhz + Fs_Mhz/2;
    
    fprintf(gnuplot, "set term wxt %d size 500,500\n", cplot++);
    fprintf(gnuplot, "set ylabel 'loss dB'; set xlabel 'freq Mhz'; set xtics ('%.1f' 1, '%.1f' %d, '%.1f' %d)\n", lowF, Fc_Mhz, n/2, highF, n-1);
    fprintf(gnuplot, "plot '-' smooth frequency with linespoints lt -1 title '%s' \n", title);
    for(int i = 0; i < n; i++){
        fprintf(gnuplot,"%d  %f\n", i, db[i]);
    }
    fprintf(gnuplot, "e\n");
    fflush(gnuplot);

}


    
void create_fft(hipFloatComplex *z, int n, int offset, hipStream_t s,
    float f_c, // freqency center 
    float f_s,  // sample rate 
    const char* title
){
    
    printf("Starting FFT\n");
    hipfftComplex *d_sig, *d_fft;
    float * d_db; 
    
    // setup data
    checkCuda( hipMalloc((void**)&d_sig, sizeof(hipfftComplex) * n) );
    checkCuda( hipMalloc((void**)&d_fft, sizeof(hipfftComplex) * n) );
    checkCuda( hipMalloc((void**)&d_db, sizeof(float) * n + 2) ); // lock and max space
    checkCuda( hipMemsetAsync(d_db, 0, sizeof(float) * n +2, s) );
    checkCuda( hipMemcpyAsync(d_sig, &z[offset], n*sizeof(hipfftComplex), hipMemcpyHostToDevice, s) );

    // setup FFT
    printf("Running FFT \n");
    hipfftHandle plan;
    checkCufft( hipfftPlan1d(&plan, n, HIPFFT_C2C, 1) ); // issuing 1 FFT of the size sample
    checkCufft( hipfftSetStream(plan, s) );
    checkCufft( hipfftExecC2C(plan, d_sig, d_fft, HIPFFT_FORWARD) ); // execute the plan
    checkCufft( hipfftDestroy(plan) ); // brick the plan after being sued

    // we have a FFT we need to normalize the db data so it makes sense
    checkCudaKernel( (findMaxC2RMag<<<2,1024, 0, s>>>(n, d_fft, d_db)) );
    checkCudaKernel( (fft2amp<<<1, 1024, 0, s>>>(n, d_fft, d_db)) );
    float * db = (float*) malloc(n*sizeof(float) + 2); 

    // db is display as  0,1,2..Fs/2 -Fs/2 ... -3 -2. -1 reorder it 
    checkCuda( hipMemcpyAsync(db, &d_db[n/2], n/2*sizeof(float),hipMemcpyDeviceToHost,s) );
    checkCuda( hipMemcpyAsync(&db[n/2], d_db, n/2*sizeof(float),hipMemcpyDeviceToHost,s) );
    checkCuda( hipStreamSynchronize(s) );

    // plot and release results
    printf("plotting fft\n");
    plotfft(f_c,f_s, n, db, title);

    printf("Free data\n");
    checkCuda( hipFree(d_sig) );
    checkCuda( hipFree(d_fft) );
    checkCuda( hipFree(d_db)  );
    free(db);
}


float* fm_demod(hipFloatComplex *signal, int *n_out, float freq_drift, float freq_sr) 
{
    // setup
    int n = *n_out;
    hipStream_t s;
    checkCuda( hipStreamCreate(&s) );
    hipFloatComplex *d_ca, *d_cb;
    float *d_ra, *d_rb;
    checkCuda( hipMalloc((void**)&d_ca, sizeof(hipFloatComplex)*n) );
    checkCuda( hipMalloc((void**)&d_cb, sizeof(hipFloatComplex)*n) );
    checkCuda( hipMalloc((void**)&d_ra, sizeof(float)*n) );
    checkCuda( hipMalloc((void**)&d_rb, sizeof(float)*n) );
    checkCuda( hipMemcpyAsync(d_ca, &signal[0], n*sizeof(hipFloatComplex), hipMemcpyHostToDevice,s) );
   
    
    printf("Shifting signal to baseband\n");
    // exec
    // center by removing drift
    checkCudaKernel( (freqShift<<<8,1024,0, s>>>(n, d_ca, freq_drift, 0, freq_sr)) );
    checkCuda( hipStreamSynchronize(s) );

    printf("Filtering at baseband 200KHz\n");
    // filter out noise
    checkCudaKernel( (blackmanFIR_200KHz<<<8,1024,0, s>>>(n, d_ca, d_cb)) );
    checkCuda( hipStreamSynchronize(s) );

    printf("Decimating signal\n");
    // Decimate to bandwidth = 200Khz
    int dec_rate = int(freq_sr / 2e5);
    float freq_sr_d1 = freq_sr / dec_rate;
    checkCudaKernel( (decimateC2C<<<8, 1024, 0, s>>>(n, dec_rate, d_ca, d_cb)));
    int n_d1 = n / dec_rate; // trunction keeps us in band
    
    checkCuda( hipStreamSynchronize(s) );
    printf("Polar discrimnate\n");
    
    // potential plot a constellation for debug
    // polar discriminate to demoulate the signal this is a C2R operation
    checkCudaKernel( (pdsC2R<<<8, 1024, 0, s>>>(n_d1, d_cb, d_ra)) );

    checkCuda( hipStreamSynchronize(s) );
    printf("Convert to audio sampler rate\n");
    
    // skiping de-emphasis fitler and just decimate to audio
    dec_rate = int(freq_sr_d1/ 44100.0); //audio samples will be at ~44.1Khz
    //float freq_sr_d2 = freq_sr_d1 / dec_rate;
    checkCudaKernel( (decimateR2R<<<8, 1024, 0, s>>>(n, dec_rate, d_ra, d_rb)) );
    int n_d2 = n_d1 / dec_rate; // stay in band

    // scale volume
    checkCuda( hipStreamSynchronize(s) );
    printf("Finding max Mag\n");
    
    checkCudaKernel( (findMaxR2RMag<<<8,1024, 0, s>>>(n_d2, d_rb, d_ra)) ); 
        // note max is stored in d_ra[n_n2] by findMaxDef
    
    checkCuda( hipStreamSynchronize(s) );
    printf("Scaling vector\n");
    
    checkCudaKernel( (scaleVec<<<8, 1024, 0, s>>>(n_d2, d_rb, d_ra[n_d2])) );
    
    checkCuda( hipStreamSynchronize(s) );
    printf("Copying data back to sig_out\n");


    *n_out = n_d2; // log the final samples count
    float *sig_out = (float*) malloc(n_d2 * sizeof(float));
    checkCuda( hipMemcpyAsync(sig_out, d_rb, n_d2*sizeof(float), hipMemcpyDeviceToHost,s) );


    printf("Cleanup\n");

    // release resources
    checkCuda( hipStreamSynchronize(s) );
    checkCuda( hipStreamDestroy(s) );
    checkCuda( hipFree(d_ca) );
    checkCuda( hipFree(d_cb) );
    checkCuda( hipFree(d_ra) );
    checkCuda( hipFree(d_rb) );

    return sig_out;
}

int main(int argc, char** argv)
{
    int n;
    double *idata, *qdata;
    hipFloatComplex *z = readData(&n, idata, qdata); // we have n complex numbers now
    free(idata); free(qdata); // unused

    #ifdef DPLOT
    gnuplot = popen("gnuplot -persistent", "w");
    #else
    gnuplot = fopen("gplot", "w"); // with live ploting off write theplots to a file
    #endif

    // Sample Rate ends up being 44Khz by convention
    printf("Running fm_demod on signal\n");
    float *audio = fm_demod(z, &n, 0.178e6, 2.5e6); 

    FILE* ad = fopen("audio.out", "w+");
    printf("Printing audio samples 2 a file\n");
    for(int i = 0; i<n; i++){
        int16_t sample = (int16_t) audio[i];
        fwrite( &sample, sizeof(sample), 1, ad);
    }
    fclose(ad);


    free(audio);   
}
