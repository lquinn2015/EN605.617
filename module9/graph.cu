#include <stdio.h>
#include <stdlib.h>
#include "cuda_utils.cuh"

#include <hip/hip_runtime.h>
#include "nvgraph.h"

void check(nvgraphStatus_t status){
    if(status != NVGRAPH_STATUS_SUCCESS){
        printf("Error : %d\n", status);
        exit(0);
    }
}


void readGraph(FILE *fp, float *val, int *dest, int *src)
{

    char *line;
    size_t amt;
    float w_i; int w_idx=0;
    int d_l=-1, d_i, d_idx = 0;
    int s_i, s_idx=0;
    
    while(getline(&line, &amt, fp) != -1){ // while we can get another line
        
        sscanf(line, "%f %d %d", &w_i, &s_i, &d_i); // parse line
        if(d_idx == 0 || d_l != d_i)
        {
            dest[d_idx++] = d_i;
            d_l = d_i;  
        }
        val[w_idx++] = w_i;
        src[s_idx++] = s_i;
    }
    return;
}


void sssp_graph(const char* fname)
{
    printf("Starting sssp\n");
    FILE* fp = fopen(fname, "r");
    size_t linesize;
    char* line;

    int len = getline(&line, &linesize, fp); // reads one line
    if(len == -1){
        exit(-1); //error
    }

    int n, nnz, ccol, vertex_numsets = 1, edge_numsets = 1;
    float *sssp_1_h;
    void **vertex_dim;
    printf("%s\n", line); 
    sscanf((const char*)line, "%d %d %d", &n, &nnz, &ccol);
    printf("Graph #vert=%d, #edges=%d\n", n, nnz);
    
    //nvgraph varibles
    nvgraphHandle_t handle;
    nvgraphGraphDescr_t graph;
    nvgraphCSCTopology32I_t CSC_input;
    hipDataType edge_dimT = HIP_R_32F;
    hipDataType* vertex_dimT;

    // init data
    sssp_1_h = (float*)malloc(n*sizeof(float));
    vertex_dim = (void**)malloc(vertex_numsets*sizeof(void*));
    vertex_dimT = (hipDataType*)malloc(vertex_numsets*sizeof(hipDataType));
    CSC_input = (nvgraphCSCTopology32I_t) malloc(sizeof(struct nvgraphCSCTopology32I_st));
    vertex_dim[0] = (void*) sssp_1_h; vertex_dimT[0] = HIP_R_32F;
    
    float *weights = (float*) malloc(nnz * sizeof(float));
    int *dest = (int*) malloc(ccol*sizeof(float));
    int *src = (int*) malloc(nnz*sizeof(float));
    readGraph(fp, weights, dest, src);
    printf("Graph IO complete running nvgraph now\n");
     
    check( nvgraphCreate(&handle));
    check( nvgraphCreateGraphDescr(handle, &graph));
    CSC_input->nvertices = n; CSC_input->nedges = nnz;
    CSC_input->destination_offsets = dest;
    CSC_input->source_indices = src;

    // Set connectivity and properties
    check( nvgraphSetGraphStructure(handle, graph, (void*)CSC_input, NVGRAPH_CSC_32));
    check( nvgraphAllocateVertexData(handle, graph, vertex_numsets, vertex_dimT));
    check( nvgraphAllocateEdgeData(handle, graph, edge_numsets, &edge_dimT));
    check( nvgraphSetEdgeData(handle, graph, (void*)weights, 0));
    
    // solve
    int src_vert = 0;
    check( nvgraphSssp(handle, graph, 0, &src_vert, 0));
    // get and print results
    check(nvgraphGetVertexData(handle, graph, (void*)sssp_1_h, 0));
    printf("Cost to get from 0->2 was %f\n", sssp_1_h[2]);

    // free data
    free(sssp_1_h); free(vertex_dim);
    free(vertex_dimT); free(CSC_input);
    check(nvgraphDestroyGraphDescr(handle, graph));
    check(nvgraphDestroy(handle));
    free(weights);    free(dest);
    free(src);     free(line);
    fclose(fp);
    return;
}

int main()
{    
    sssp_graph("csc1.lsv");
    sssp_graph("csc2.lsv");
    sssp_graph("csc3.lsv");
    return 0;
}
