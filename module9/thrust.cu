#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <iostream>
#include <stdlib.h>
#include "cuda_utils.cuh"
#include <argp.h>


static int problem_size = 0;
static int blocksize;

static int parse_opt(int key, char *arg, struct argp_state *state)
{

    if( arg == NULL) {
        return 0;
    }

    switch (key) {
        
        case 'p': {
            problem_size = atoi(arg);
            printf("n = %d\n", problem_size);
            break; 
        } case 'b' : {
            blocksize = atoi(arg);
            printf("blocksize = %d\n", blocksize);
            break;
        }

    }
    return 0;
}

struct argp_option options[] = 
{
    {"size", 'p', "NUM", OPTION_ARG_OPTIONAL, "Problem size to work on"},
    {"bsize", 'b', "NUM", OPTION_ARG_OPTIONAL, "Block size"},
    { 0 }
};

void hello(int n) {

    thrust::host_vector<int> H(n);
    H[0] = 1;
    H[4] = 4;
    std::cout << "H[4]=" << H[4] << " overall H has len=" << H.size() << std::endl; 

    thrust::device_vector<int> D = H;
    D[4] = 8;
    std::cout << "D[0]=" << D[0] << " overall D has len=" << D.size() << std::endl; 
    std::cout << "D[4]=" << D[4] << " overall D has len=" << D.size() << std::endl; 

    thrust::device_vector<int> C(n);
    C[4] = 9;
    std::cout << "C[4]=" << C[4] << " overall C has len=" << C.size() << std::endl; 

    thrust::generate(thrust::host, H.begin(), H.end(), rand);
    std::cout << "H[4]=" << H[4] << " overall H has len=" << H.size() << std::endl; 
    
    thrust::generate(thrust::device, D.begin(), D.end(), rand);
    std::cout << "D[0]=" << D[0] << " overall D has len=" << D.size() << std::endl; 
    std::cout << "D[4]=" << D[4] << " overall D has len=" << D.size() << std::endl; 

    return;

}

void basicThrustTest(int n){

    // Given X,Y   compute  X = (X^2 + X - Y) % Y
    double start = clock();
    std::cout << "Thrust slow compute test\n" << std::endl;

    thrust::host_vector<int> H(n);

    thrust::generate(H.begin(), H.end(), rand);
    thrust::device_vector<int> X = H;

    thrust::generate(H.begin(), H.end(), rand);
    thrust::device_vector<int> Y = H;

    thrust::device_vector<int> Z(n);
    
    int sel = rand() % n; 
    thrust::generate(X.begin(), X.end(), rand);
    thrust::generate(Y.begin(), Y.end(), rand);
   
    std::cout << "X[" << sel << "] = " << X[sel] << std::endl;
    std::cout << "Y[" << sel << "] = " << Y[sel] << std::endl;
 
    // Z = X*X
    thrust::transform(X.begin(), X.end(), 
        X.begin(), 
        Z.begin(),  
        thrust::multiplies<int>()
    );

    // Z = Z + X
    thrust::transform(Z.begin(), Z.end(), 
        X.begin(), 
        Z.begin(),  
        thrust::plus<int>()
    );
    // Z = Z - Y
    thrust::transform(Z.begin(), Z.end(), 
        Y.begin(), 
        Z.begin(),  
        thrust::minus<int>()
    );
    // Z = Z % Y
    thrust::transform(Z.begin(), Z.end(), 
        Y.begin(), 
        Z.begin(),  
        thrust::modulus<int>()
    );

    std::cout << "Z[" << sel << "] = " << Z[sel] << std::endl; 
 
    double diff = clock() - start;
    std::cout << "basic test Time spent " << diff << std::endl;
    
    return;
}

struct fast_functor {
    
    __host__ __device__
        float operator()(const int &x, const int &y) const{
            return (x*x + x - y ) % y;
        }
};

void compoundThrustTest(int n){

    // Given X,Y   compute   (X^2 + X - Y) % Y
    double start = clock();
    std::cout << "Thrust fast compute test\n" << std::endl;

    thrust::device_vector<int> X(n);
    thrust::device_vector<int> Y(n);
    thrust::device_vector<int> Z(n);
    
    thrust::generate(X.begin(), X.end(), rand);
    thrust::generate(Y.begin(), Y.end(), rand);

    thrust::transform(X.begin(), X.end(),
        Y.begin(),
        Z.begin(),
        fast_functor()
    );
    
    int sel = rand() % n; 
    std::cout << "Z[" << sel << "] = " << Z[sel] << std::endl; 
    
    double diff = clock() - start;
    std::cout << "fast test Time spent " << diff << std::endl;

}

int main(int argc, char **argv){

    struct argp argp = {options, parse_opt, 0, 0};
    argp_parse(&argp, argc, argv, 0, 0, 0);
    srand(time(NULL));
    int n = problem_size;

    hello(n);
    basicThrustTest(n);
    compoundThrustTest(n);

    return 0;
}
